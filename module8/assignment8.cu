#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <stdint.h>
#include <assert.h>
#include <time.h>
#include <math.h>

/* we need these includes for CUDA's random number stuff */
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#include "hipblas.h"

#include "hip/hip_runtime.h"
#include ""
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

/*
Author: Andrew DiPrinzio 
Course: EN605.417.FA
Assignment: Module 8
Resources: 
http://docs.nvidia.com/cuda/hiprand/host-api-overview.html
https://developer.nvidia.com/sites/default/files/akamai/cuda/files/Misc/mygpu.pdf
*/


static void usage(){    
    printf("Usage: ./assignment8 [-h]\n");
}

// Parse the command line arguments using getopt and return an Argument structure
// GetOpt requies the POSIX C Library
void parse_arguments(const int argc, char ** argv){   
    // Argument format string for getopt
    static const char * _ARG_STR = "h";
    // Initialize arguments to their default values    
    // Arguments args;    
    // args.num_threads = DEFAULT_NUM_THREADS;    
    // args.block_size = DEFAULT_BLOCK_SIZE;
    // Parse any command line options
    int c;
    //int value;
    while ((c = getopt(argc, argv, _ARG_STR)) != -1) {
        switch (c) {
            case 'h':
                // 'help': print usage, then exit
                // note the fall through
                usage();
            default:
                exit(-1);
        }
    }
}

/* ******************* CURAND Section *******************/

#define MAX_RAND 100
 
#define N_RAND 100

/* this GPU kernel function is used to initialize the random states */
__global__
void init(unsigned int seed, hiprandState_t* states)
{
    /* we have to initialize the state */
    hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
                blockIdx.x, /* the sequence number should be different for each core (unless you want all
                                cores to get the same sequence of numbers for some reason - use thread id! */
                0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
                &states[blockIdx.x]);
}
 
/* this GPU kernel takes an array of states, and an array of ints, and puts a random int into each */
__global__
void randoms(hiprandState_t* states, unsigned int* numbers)
{
    /* hiprand works like rand - except that it takes a state as a parameter */
    numbers[blockIdx.x] = hiprand(&states[blockIdx.x]) % 100;
}

__host__
void run_cuda_rand_kernel()
{
    // create events for timing
    hipEvent_t startEvent, stopEvent; 
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    /* CUDA's random number library uses hiprandState_t to keep track of the seed value
    we will store a random state for every thread  */
    hiprandState_t* states;

    /* allocate space on the GPU for the random states */
    hipMalloc((void**) &states, N_RAND * sizeof(hiprandState_t));

    /* invoke the GPU to initialize all of the random states */
    init<<<N_RAND, 1>>>(time(0), states);

    /* allocate an array of unsigned ints on the CPU and GPU */
    unsigned int cpu_nums[N_RAND];
    unsigned int* gpu_nums;
    hipMalloc((void**) &gpu_nums, N_RAND * sizeof(unsigned int));

    /* invoke the kernel to get some random numbers */
    hipEventRecord(startEvent, 0);
    randoms<<<N_RAND, 1>>>(states, gpu_nums);
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);

    /* copy the random numbers back */
    hipMemcpy(cpu_nums, gpu_nums, N_RAND * sizeof(unsigned int), hipMemcpyDeviceToHost);

    /* print them out */
    //   for (int i = 0; i < N_RAND; i++) {
    //     printf("%u\n", cpu_nums[i]);
    //   }

    float time;
    hipEventElapsedTime(&time, startEvent, stopEvent);
    printf("cuRAND deviceAPI Exec Time: %f ms\n", time);

    /* free the memory we allocated for the states and numbers */
    hipFree(states);
    hipFree(gpu_nums);
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
}

void run_cuda_rand_hostAPI( ) {
    size_t n = N_RAND;
    
    hiprandGenerator_t gen;

    // create events for timing
    hipEvent_t startEvent, stopEvent; 
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    float * devData, * hostData; /* Allocate n floats on host */

    hostData = (float * ) calloc(n, sizeof(float)); /* Allocate n floats on device */

    hipMalloc((void * * ) & devData, n * sizeof(float)); /* Create pseudo-random number generator */
    hiprandCreateGenerator( & gen, HIPRAND_RNG_PSEUDO_DEFAULT); /* Set seed */

    hipEventRecord(startEvent, 0);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL); /* Generate n floats on device */
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);

    hiprandGenerateUniform(gen, devData, n); /* Copy device memory to host */
    hipMemcpy(hostData, devData, n * sizeof(float), hipMemcpyDeviceToHost); /* Show result */

    float time;
    hipEventElapsedTime(&time, startEvent, stopEvent);
    printf("cuRAND HostAPI Exec Time: %f ms\n", time);
    
    // size_t i;
    // for (i = 0; i < n; i++) {
    //     printf("%1.4f ", hostData[i]);
    // }
    
    printf("\n"); /* Cleanup */
    hiprandDestroyGenerator(gen);
    hipFree(devData);
    free(hostData);
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
}

/* ******************* CUBLAS Section *******************/

// Helper function to generate a random number within a defined range
float random(int max){
    return  (float)rand()/(float)(RAND_MAX/max);
}

void run_cuBLAS_saxpy() {
    // create events for timing
    hipEvent_t startEvent, stopEvent; 
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    hipblasHandle_t handle; // CUBLAS context

    int array_size = 320000;
    const unsigned int array_size_in_bytes = array_size * sizeof(int);

    /* Randomly generate input vectors and dynamically allocate their memory */
    float * x; 
    float * y;
    
    x = (float*)malloc(array_size * sizeof(float));
    y = (float*)malloc(array_size * sizeof(float));

    int i;
    for (i = 0; i < array_size; i++) {
        x[i] = random(100);
    }
    for (i = 0; i < array_size; i++) {
        y[i] = random(100);
    }

    /* Declare pointers for GPU based params */
    float *x_d;
    float *y_d;

    hipMalloc((void**)&x_d, array_size_in_bytes);
    hipMalloc((void**)&y_d, array_size_in_bytes);
    hipMemcpy( x_d, x, array_size_in_bytes, hipMemcpyHostToDevice );
    hipMemcpy( y_d, y, array_size_in_bytes, hipMemcpyHostToDevice );

    hipblasCreate(&handle); // initialize CUBLAS context
    hipblasSetVector(array_size, sizeof( *x), x, 1, x_d, 1); // cp x- >x_d
    hipblasSetVector(array_size, sizeof( *y), y, 1, y_d, 1); // cp y- >y_d
    float al = 2.0; // al =2
    // multiply the vector x_d by the scalar al and add to y_d
    // y_d = al*x_d + y_d , x_d ,y_d - n- vectors ; al - scalar
    hipEventRecord(startEvent, 0);
    hipblasSaxpy(handle, array_size, &al, x_d, 1, y_d, 1);
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);

    hipblasGetVector(array_size, sizeof(float), y_d, 1, y, 1); // cp y_d - >y

    float time;
    hipEventElapsedTime(&time, startEvent, stopEvent);
    printf("cuBLAS Saxpy Exec Time: %f ms\n", time);

    // printf("y after Saxpy :\n"); // print y after Saxpy
    // for (j = 0; j < array_size; j++)
    //     printf(" %2.0f,", y[j]);
    
    printf("\n");
    hipFree(x_d); // free device memory
    hipFree(y_d); // free device memory
    hipblasDestroy(handle); // destroy CUBLAS context
    free(x); // free host memory
    free(y); // free host memory
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
}

//Kernel that performs saxpy
__global__
void custom_saxpy(const float * a , const float *x, float *y)
{
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	y[thread_idx] = (*a) * x[thread_idx] + y[thread_idx] ;
}

void run_custom_saxpy(){
    printf("Running custom_saxpy\n");
    int array_size = 320000;
    const unsigned int array_size_in_bytes = array_size * sizeof(int);

    // create events for timing
    hipEvent_t startEvent, stopEvent; 
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    /* Randomly generate input vectors and dynamically allocate their memory */
    float * x; 
    float * y;
   	float * a;
    
    x = (float*)malloc(array_size * sizeof(float));
    y = (float*)malloc(array_size * sizeof(float));
    a = (float*)malloc(sizeof(float));

    int i;
    for (i = 0; i < array_size; i++) {
        x[i] = random(100);
    }
    for (i = 0; i < array_size; i++) {
        y[i] = random(100);
    }
	a[0] = (float) 2;

	/* Declare pointers for GPU based params */
	float *x_d;
    float *y_d;
    float *a_d;

    hipMalloc((void**)&x_d, array_size_in_bytes);
    hipMalloc((void**)&y_d, array_size_in_bytes);
    hipMalloc((void**)&a_d, sizeof(float));
	hipMemcpy( x_d, x, array_size_in_bytes, hipMemcpyHostToDevice );
    hipMemcpy( y_d, y, array_size_in_bytes, hipMemcpyHostToDevice );
    hipMemcpy( a_d, a, sizeof(float), hipMemcpyHostToDevice );

	const unsigned int num_blocks = array_size / 32;
    const unsigned int num_threads_per_blk = array_size/num_blocks;
    
    hipEventRecord(startEvent, 0);
	/* Execute our kernel */
	custom_saxpy<<<num_blocks, num_threads_per_blk>>>(a_d,x_d,y_d);
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);

    float time;
    hipEventElapsedTime(&time, startEvent, stopEvent);
    printf("Custom Saxpy Exec Time: %f ms\n", time);

	/* Free the arrays on the GPU as now we're done with them */
    hipMemcpy(y, y_d, array_size_in_bytes, hipMemcpyDeviceToHost );
    
	hipFree(x_d);
    hipFree(y_d);
    hipFree(a_d);
    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);
}

int main(int argc, char ** argv)
{
    parse_arguments(argc, argv);

    printf("Runing cuRAND Experements \n");

    printf("cuRAND device API \n");
    run_cuda_rand_kernel();

    printf("\ncuRAND host API \n");
    run_cuda_rand_hostAPI();

    printf("Runing cuBLAS Experement\n");
    run_cuBLAS_saxpy();

    run_custom_saxpy();

	return EXIT_SUCCESS;
}