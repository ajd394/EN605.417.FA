
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <stdint.h>
#include <assert.h>
#include <time.h>
#include <math.h>

/*
Author: Andrew DiPrinzio 
Course: EN605.417.FA
*/

static const uint32_t DEFAULT_NUM_THREADS = 1024;
static const uint32_t DEFAULT_BLOCK_SIZE = 16;

static void usage(){    
    printf("Usage: ./assignment3 [-t <num_threads>] [-b <block_size>] [-h]\n");
   
    printf("\t-t: Specify the number of threads. <num_threads> must be greater than 0. Optional (default %u)\n", DEFAULT_NUM_THREADS);
   
    printf("\t-b: Specify the size of each block. <block_size> must be greater than 0. Optional (default %u)\n", DEFAULT_BLOCK_SIZE);    
}

// Structure that holds program arguments specifying number of threads/blocks
// to use.
typedef struct {    
    uint32_t num_threads;
    uint32_t block_size;
} Arguments;

// Parse the command line arguments using getopt and return an Argument structure
// GetOpt requies the POSIX C Library
static Arguments parse_arguments(const int argc, char ** argv){   
    // Argument format string for getopt
    static const char * _ARG_STR = "ht:b:";
    // Initialize arguments to their default values    
    Arguments args;    
    args.num_threads = DEFAULT_NUM_THREADS;    
    args.block_size = DEFAULT_BLOCK_SIZE;
    // Parse any command line options
    int c;
    int value;
    while ((c = getopt(argc, argv, _ARG_STR)) != -1) {
        switch (c) {
            case 't':
                value = atoi(optarg);
                args.num_threads = value;
                break;
            case 'b':
                // Normal argument
                value = atoi(optarg);
                args.block_size = value;
                break;
            case 'h':
                // 'help': print usage, then exit
                // note the fall through
                usage();
            default:
                exit(-1);
        }
    }
    return args;
}

//Kernel that adds two vectors
__global__
void add_ab(int *a, const int *b)
{
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	a[thread_idx] += b[thread_idx];
}

// Helper function to generate a random number within a defined range
int random(int min, int max){
    return min + rand() / (RAND_MAX / (max - min + 1) + 1);
}

void run_vector_add(Arguments args)
{ 
    printf("Running random vector add with %u threads and a block size of %u\n", args.num_threads, args.block_size);
    int array_size = args.num_threads;
    const unsigned int array_size_in_bytes = array_size * sizeof(int);

    /* Randomly generate input vectors and dynamically allocate their memory */
    int * a; 
    int * b;
    
    a = (int*)malloc(array_size * sizeof(int));
    b = (int*)malloc(array_size * sizeof(int));

    int i;
    for (i = 0; i < array_size; i++) {
        a[i] = random(0,100);
    }
    for (i = 0; i < array_size; i++) {
        b[i] = random(0,100);
    }

	/* Declare pointers for GPU based params */
    int *a_d;
	int *b_d;

	hipMalloc((void**)&a_d, array_size_in_bytes);
	hipMalloc((void**)&b_d, array_size_in_bytes);
	hipMemcpy( a_d, a, array_size_in_bytes, hipMemcpyHostToDevice );
	hipMemcpy( b_d, b, array_size_in_bytes, hipMemcpyHostToDevice );

	const unsigned int num_blocks = array_size / args.block_size;
	const unsigned int num_threads_per_blk = array_size/num_blocks;

	/* Execute our kernel */
	add_ab<<<num_blocks, num_threads_per_blk>>>(a_d, b_d);

	/* Free the arrays on the GPU as now we're done with them */
	hipMemcpy(a, a_d, array_size_in_bytes, hipMemcpyDeviceToHost );
	hipFree(a_d);
	hipFree(b_d);

	/* Iterate through the result array and print */
    for(unsigned int i = 0; i < array_size; i++)
	{
		printf("Sum #%d: %d\n",i,a[i]);
	}
}

int main(int argc, char ** argv)
{
    Arguments args = parse_arguments(argc, argv);

    run_vector_add(args);
    
	return EXIT_SUCCESS;
}