/*
Author: Andrew DiPrinzio 
Course: EN605.417.FA
Assignment: Module 5
*/


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <stdint.h>
#include <assert.h>
#include <time.h>
#include <math.h>

static const uint32_t DEFAULT_NUM_THREADS = 1024;
static const uint32_t DEFAULT_BLOCK_SIZE = 16;

#define KERNEL_LOOP 4096

__constant__ unsigned int const_data_gpu[KERNEL_LOOP];
__device__ static unsigned int gmem_data_gpu[KERNEL_LOOP];
static unsigned int const_data_host[KERNEL_LOOP];

static void usage(){    
    printf("Usage: ./assignment5 [-t <num_threads>] [-b <block_size>] [-h]\n");
   
    printf("\t-t: Specify the number of threads. <num_threads> must be greater than 0. Optional (default %u)\n", DEFAULT_NUM_THREADS);
   
    printf("\t-b: Specify the size of each block. <block_size> must be greater than 0. Optional (default %u)\n", DEFAULT_BLOCK_SIZE);    
}
    

// Structure that holds program arguments specifying number of threads/blocks
// to use.
typedef struct {    
    uint32_t num_threads;
    uint32_t block_size;
} Arguments;

// Parse the command line arguments using getopt and return an Argument structure
// GetOpt requies the POSIX C Library
static Arguments parse_arguments(const int argc, char ** argv){   
    // Argument format string for getopt
    static const char * _ARG_STR = "ht:b:";
    // Initialize arguments to their default values    
    Arguments args;    
    args.num_threads = DEFAULT_NUM_THREADS;    
    args.block_size = DEFAULT_BLOCK_SIZE;
    // Parse any command line options
    int c;
    int value;
    while ((c = getopt(argc, argv, _ARG_STR)) != -1) {
        switch (c) {
            case 't':
                value = atoi(optarg);
                args.num_threads = value;
                break;
            case 'b':
                // Normal argument
                value = atoi(optarg);
                args.block_size = value;
                break;
            case 'h':
                // 'help': print usage, then exit
                // note the fall through
                usage();
            default:
                exit(-1);
        }
    }
    return args;
}

__global__ void const_test_gpu_gmem(unsigned int * const data, const unsigned int num_elements)
{
	const unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(tid < num_elements)
	{
		unsigned int d = gmem_data_gpu[0];

		for(int i=0;i<KERNEL_LOOP;i++)
		{
			d ^= gmem_data_gpu[0];
			d |= gmem_data_gpu[1];
			d &= gmem_data_gpu[2];
			d |= gmem_data_gpu[3];
		}

		data[tid] = d;
	}
}


__global__ void const_test_gpu_const(unsigned int * const data, const unsigned int num_elements)
{
	const unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(tid < num_elements)
	{
		unsigned int d = const_data_gpu[0];

		for(int i=0;i<KERNEL_LOOP;i++)
		{
			d ^= const_data_gpu[0];
			d |= const_data_gpu[1];
			d &= const_data_gpu[2];
			d |= const_data_gpu[3];
		}

		data[tid] = d;
	}
}

__host__ void wait_exit(void)
{
	char ch;

	printf("\nPress any key to exit");
	ch = getchar();
}

__host__ void cuda_error_check(const char * prefix, const char * postfix)
{
	if(hipPeekAtLastError() != hipSuccess)
	{
		printf("\n%s%s%s",prefix,hipGetErrorString(hipGetLastError()),postfix);
		hipDeviceReset();
		wait_exit();
		exit(1);
	}
}

__host__ void generate_rand_data(unsigned int * host_data_ptr)
{
	for(unsigned int i=0; i < KERNEL_LOOP; i++)
	{
		host_data_ptr[i] = (unsigned int) rand();
	}
}
__host__ void test_const_mem(Arguments args)
{
	const unsigned int num_elements = (128*1024);
	const unsigned int num_threads = args.num_threads;
	const unsigned int num_blocks = (num_elements + (num_threads-1))/num_threads;
	const unsigned int num_bytes = num_elements * sizeof(unsigned int);
	int max_device_num;
	const int max_runs = 6;

	hipGetDeviceCount(&max_device_num);

	for(int device_num=0; device_num < max_device_num; device_num++)
	{
		hipSetDevice(device_num);

		unsigned int * data_gpu;
		hipEvent_t kernel_start1, kernel_stop1;
		hipEvent_t kernel_start2, kernel_stop2;
		float delta_time1 = 0.0F, delta_time2 = 0.0F;
		struct hipDeviceProp_t device_prop;
		char device_prefix[261];

		hipMalloc(&data_gpu, num_bytes);
		hipEventCreate(&kernel_start1);
		hipEventCreate(&kernel_start2);
		hipEventCreateWithFlags(&kernel_stop1, hipEventBlockingSync);
		hipEventCreateWithFlags(&kernel_stop2, hipEventBlockingSync);

		hipGetDeviceProperties(&device_prop, device_num);
		sprintf(device_prefix, "ID: %d %s:", device_num, device_prop.name);

		for(int num_test=0; num_test < max_runs; num_test++)
		{
			generate_rand_data(const_data_host);

			hipMemcpyToSymbol(HIP_SYMBOL(const_data_gpu), const_data_host, KERNEL_LOOP * sizeof(unsigned int));

			const_test_gpu_gmem <<<num_blocks, num_threads>>>(data_gpu, num_elements);
			cuda_error_check("Error ", " returned from literal runtime  kernel!");

			hipEventRecord(kernel_start1,0);

			const_test_gpu_gmem <<<num_blocks, num_threads>>>(data_gpu, num_elements);

			cuda_error_check("Error ", " returned from literal runtime  kernel!");

			hipEventRecord(kernel_stop1,0);
			hipEventSynchronize(kernel_stop1);
			hipEventElapsedTime(&delta_time1, kernel_start1, kernel_stop1);

			hipMemcpyToSymbol(HIP_SYMBOL(gmem_data_gpu), const_data_host, KERNEL_LOOP * sizeof(unsigned int));
			const_test_gpu_const<<< num_blocks, num_threads >>>(data_gpu, num_elements);

			cuda_error_check("Error ", " returned from literal startup  kernel!");

			hipEventRecord(kernel_start2,0);

			const_test_gpu_const<<< num_blocks, num_threads >>>(data_gpu, num_elements);

			cuda_error_check("Error ", " returned from literal startup  kernel!");

			hipEventRecord(kernel_stop2,0);
			hipEventSynchronize(kernel_stop2);
			hipEventElapsedTime(&delta_time2, kernel_start2, kernel_stop2);

			if(delta_time1 > delta_time2)
			{
				printf("\n%sConstant version is faster by: %.2fms (G=%.2fms vs. C=%.2fms)",device_prefix, delta_time1-delta_time2, delta_time1, delta_time2);
			}
			else
			{
				printf("\n%sGMEM version is faster by: %.2fms (G=%.2fms vs. C=%.2fms)",device_prefix, delta_time2-delta_time1, delta_time1, delta_time2);
			}

		}

		hipEventDestroy(kernel_start1);
		hipEventDestroy(kernel_start2);
		hipEventDestroy(kernel_stop1);
		hipEventDestroy(kernel_stop2);
		hipFree(data_gpu);

		hipDeviceReset();
		printf("\n");
	}
	wait_exit();
}

int main(int argc, char ** argv)
{
    Arguments args = parse_arguments(argc, argv);
    printf("Num Threads: %u, Block Size: %u\n", args.num_threads, args.block_size);

    test_const_mem(args);
    
	return EXIT_SUCCESS;
}
